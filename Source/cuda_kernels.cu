#include "hip/hip_runtime.h"
#ifndef __CUDACC_RTC__

#include "../Include/objects_list.h"
#include "../Include/cuda_defines.cuh"

#else

#define CREATE_OBJECT_TYPE_DESCRIPTION(__TYPE__,__STRUCT__)                                         \
class __TYPE__ {                                                                                    \
protected:                                                                                          \
public:                                                                                             \
    typedef __STRUCT__ data_struct;                                                                 \
};

#include "objects_list.h"
#include "cuda_defines.cuh"

#endif

#define _ATOMIC             __device__ __forceinline__
#define _KERNEL             extern "C" __global__ void
#define _PTR                *__restrict__

namespace atomic {

_ATOMIC point   new_point( const scalar &x, const scalar &y, const scalar &z ) {
    return point{ x, y, z };
}

_ATOMIC point   mul_point( const point &p, const scalar &s ) {
    return point{ s * p.x, s * p.y, s * p.z };
}

_ATOMIC point   add_point( const point &p1, const point &p2 ) {
    return point{ p1.x + p2.x, p1.y + p2.y, p1.z + p2.z };
}

_ATOMIC scalar  dot( const point &p1, const point &p2 ) {
    return p1.x * p2.x + p1.y * p2.y + p1.z * p2.z;
}

_ATOMIC scalar  mix( const scalar &a, const scalar &b, const scalar &x ) {
    return a + x * ( b - a );
}

};

namespace primitives {

// TYPE_LIST
CREATE_OBJECT_TYPE_DEFINITION(
        portanta_sfero,
        {
            point P;
            P.x = p.x - data->t.x;
            P.y = p.y - data->t.y;
            P.z = p.z - data->t.z;
            scalar               d = length_3( P.x, P.y, P.z ) - data->r;
            primitives::bazo_ptr o = obj + data->o;
            if ( d <= RAYS_MIN_DIST ) return RAYS_DIST( o, P );
            else return d;
        },
        {
            point P;
            P.x = p.x - data->t.x;
            P.y = p.y - data->t.y;
            P.z = p.z - data->t.z;
            primitives::bazo_ptr o = obj + data->o;
            return RAYS_NORM( o, P );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        sfero,
        {
            return length_3( p.x, p.y, p.z ) - data->r;
        },
        {
            return p;
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        kubo,
        {
            point q;
            q.x = fabsf( p.x ) - data->b.x;
            q.y = fabsf( p.y ) - data->b.y;
            q.z = fabsf( p.z ) - data->b.z;
            if ( q.x < 0.f && q.y < 0.f && q.z < 0.f )
                return max( q.x, max( q.y, q.z ) );
            else
                return length_3( max( q.x, 0.f ), max( q.y, 0.f ), max( q.z, 0.f ) );
        },
        {
            point q;
            q.x = fabsf( p.x ) - data->b.x;
            q.y = fabsf( p.y ) - data->b.y;
            q.z = fabsf( p.z ) - data->b.z;
            return q.x > q.z ? ( q.x > q.y ? atomic::new_point( p.x > 0.f ? 1.f : -1.f, 0.f, 0.f ) : atomic::new_point(
                    0.f, p.y > 0.f ? 1.f : -1.f, 0.f ) ) : ( q.y > q.z ? atomic::new_point( 0.f, p.y > 0.f ? 1.f : -1.f,
                                                                                            0.f ) : atomic::new_point(
                    0.f, 0.f, p.z > 0.f ? 1.f : -1.f ) );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        cilindro,
        {
            scalar r = length_2( p.x, p.y );
            float2 q;
            q.x = r - data->r;
            q.y = fabsf( p.z ) - data->h;
            if ( q.x < 0.f && q.y < 0.f )
                return q.x > q.y ? q.x : q.y;
            else
                return length_2( max( q.x, 0.f ), max( q.y, 0.f ) );
        },
        {
            scalar r = length_2( p.x, p.y );
            float2 q;
            q.x      = r - data->r;
            q.y      = fabsf( p.z ) - data->h;
            
            return q.x > q.y ? atomic::new_point( p.x, p.y, 0.f ) : atomic::new_point( 0.f, 0.f,
                                                                                       p.z > 0.f ? 1.f : -1.f );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        ebeno,
        {
            return atomic::dot( data->n, p );
        },
        {
            return data->n;
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        kunigajo_2,
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            return min( d0, d1 );
        },
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            
            if ( d0 < d1 ) return RAYS_NORM( o0, p );
            else return RAYS_NORM( o1, p );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        kunigajo_3,
        {
            primitives::bazo_ptr o = obj + data->o[ 0 ];
            scalar               d = RAYS_DIST( o, p );
            
            o = obj + data->o[ 1 ];
            d = min( d, RAYS_DIST( o, p ) );
            
            o = obj + data->o[ 2 ];
            return min( d, RAYS_DIST( o, p ) );
        },
        {
            counter              i_min = 0;
            primitives::bazo_ptr o     = obj + data->o[ 0 ];
            scalar               d;
            scalar               d_min = RAYS_DIST( o, p );
            
            o = obj + data->o[ 1 ];
            d = RAYS_DIST( o, p );
            if ( d_min > d ) {
                d_min = d;
                i_min = 1;
            }
            
            o = obj + data->o[ 2 ];
            d = RAYS_DIST( o, p );
            if ( d_min > d ) { i_min = 2; }
            
            o = obj + data->o[ i_min ];
            return RAYS_NORM( o, p );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        kunigajo_4,
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            return min( d0, d1 );
        },
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            
            if ( d0 < d1 ) return RAYS_NORM( o0, p );
            else return RAYS_NORM( o1, p );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        komunajo_2,
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            return max( d0, d1 );
        },
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            
            if ( d0 > d1 ) return RAYS_NORM( o0, p );
            else return RAYS_NORM( o1, p );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        komunajo_3,
        {
            primitives::bazo_ptr o = obj + data->o[ 0 ];
            scalar               d = RAYS_DIST( o, p );
            
            o = obj + data->o[ 1 ];
            d = max( d, RAYS_DIST( o, p ) );
            
            o = obj + data->o[ 2 ];
            return max( d, RAYS_DIST( o, p ) );
        },
        {
            counter              i_max = 0;
            primitives::bazo_ptr o     = obj + data->o[ 0 ];
            scalar               d;
            scalar               d_max = RAYS_DIST( o, p );
            
            o = obj + data->o[ 1 ];
            d = RAYS_DIST( o, p );
            if ( d_max < d ) {
                d_max = d;
                i_max = 1;
            }
            
            o = obj + data->o[ 2 ];
            d = RAYS_DIST( o, p );
            if ( d_max < d ) { i_max = 2; }
            
            o = obj + data->o[ i_max ];
            return RAYS_NORM( o, p );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        komunajo_4,
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            return max( d0, d1 );
        },
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            
            if ( d0 > d1 ) return RAYS_NORM( o0, p );
            else return RAYS_NORM( o1, p );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        komplemento,
        {
            primitives::bazo_ptr O = obj + data->o;
            scalar               D = RAYS_DIST( O, p );
            return -D;
        },
        {
            primitives::bazo_ptr O = obj + data->o;
            point                N = RAYS_NORM( O, p );
            return atomic::new_point( -N.x, -N.y, -N.z );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        glata_kunigajo_2,
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            scalar               h  = ( 1.f - ( d0 - d1 ) / data->k ) * .5f;
            if ( h > 1.f ) return d0;
            if ( h < 0.f ) return d1;
            return atomic::mix( d0, d1, h ) - data->k * h * ( 1.f - h );
        },
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            scalar               h  = ( 1.f - ( d0 - d1 ) / data->k ) * .5f;
            if ( h > 1.f ) return RAYS_NORM( o0, p );
            if ( h < 0.f ) return RAYS_NORM( o1, p );
            point n0 = RAYS_NORM( o0, p );
            point n1 = RAYS_NORM( o1, p );
            d0 = r_length_3( n0.x, n0.y, n0.z );
            d1 = r_length_3( n1.x, n1.y, n1.z );
            return atomic::new_point( atomic::mix( d0 * n0.x, d1 * n1.x, h ), atomic::mix( d0 * n0.y, d1 * n1.y, h ),
                                      atomic::mix( d0 * n0.z, d1 * n1.z, h ) );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        glata_komunajo_2,
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            scalar               h  = ( 1.f + ( d0 - d1 ) / data->k ) * .5f;
            if ( h > 1.f ) return d0;
            if ( h < 0.f ) return d1;
            return atomic::mix( d0, d1, h ) + data->k * h * ( 1.f - h );
        },
        {
            primitives::bazo_ptr o0 = obj + data->o[ 0 ];
            primitives::bazo_ptr o1 = obj + data->o[ 1 ];
            scalar               d0 = RAYS_DIST( o0, p );
            scalar               d1 = RAYS_DIST( o1, p );
            scalar               h  = ( 1.f + ( d0 - d1 ) / data->k ) * .5f;
            if ( h > 1.f ) return RAYS_NORM( o0, p );
            if ( h < 0.f ) return RAYS_NORM( o1, p );
            point n0 = RAYS_NORM( o0, p );
            point n1 = RAYS_NORM( o1, p );
            d0 = r_length_3( n0.x, n0.y, n0.z );
            d1 = r_length_3( n1.x, n1.y, n1.z );
            return atomic::new_point( atomic::mix( d0 * n0.x, d1 * n1.x, h ), atomic::mix( d0 * n0.y, d1 * n1.y, h ),
                                      atomic::mix( d0 * n0.z, d1 * n1.z, h ) );
        } );


CREATE_OBJECT_TYPE_DEFINITION(
        movo,
        {
            primitives::bazo_ptr O = obj + data->o;
            point                P;
            P.x = p.x - data->t.x;
            P.y = p.y - data->t.y;
            P.z = p.z - data->t.z;
            return RAYS_DIST( O, P );
        },
        {
            primitives::bazo_ptr O = obj + data->o;
            point                P;
            P.x                    = p.x - data->t.x;
            P.y                    = p.y - data->t.y;
            P.z                    = p.z - data->t.z;
            return RAYS_NORM( O, P );
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        rotacioX,
        {
            primitives::bazo_ptr O = obj + data->o;
            point                P;
            P.y = data->cos_phi * p.y + data->sin_phi * p.z;
            P.z = -data->sin_phi * p.y + data->cos_phi * p.z;
            P.x = p.x;
            return RAYS_DIST( O, P );
        },
        {
            primitives::bazo_ptr O = obj + data->o;
            point                P;
            point                _P;
            P.y                    = data->cos_phi * p.y + data->sin_phi * p.z;
            P.z                    = -data->sin_phi * p.y + data->cos_phi * p.z;
            P.x                    = p.x;
            _P = RAYS_NORM( O, P );
            P.y = data->cos_phi * _P.y - data->sin_phi * _P.z;
            P.z = data->sin_phi * _P.y + data->cos_phi * _P.z;
            P.x = _P.x;
            return P;
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        rotacioY,
        {
            primitives::bazo_ptr O = obj + data->o;
            point                P;
            P.z = data->cos_phi * p.z + data->sin_phi * p.x;
            P.x = -data->sin_phi * p.z + data->cos_phi * p.x;
            P.y = p.y;
            return RAYS_DIST( O, P );
        },
        {
            primitives::bazo_ptr O = obj + data->o;
            point                P;
            point                _P;
            P.z                    = data->cos_phi * p.z + data->sin_phi * p.x;
            P.x                    = -data->sin_phi * p.z + data->cos_phi * p.x;
            P.y                    = p.y;
            _P = RAYS_NORM( O, P );
            P.z = data->cos_phi * _P.z - data->sin_phi * _P.x;
            P.x = data->sin_phi * _P.z + data->cos_phi * _P.x;
            P.y = _P.y;
            return P;
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        rotacioZ,
        {
            primitives::bazo_ptr O = obj + data->o;
            point                P;
            P.x = data->cos_phi * p.x + data->sin_phi * p.y;
            P.y = -data->sin_phi * p.x + data->cos_phi * p.y;
            P.z = p.z;
            return RAYS_DIST( O, P );
        },
        {
            primitives::bazo_ptr O = obj + data->o;
            point                P;
            point                _P;
            P.x                    = data->cos_phi * p.x + data->sin_phi * p.y;
            P.y                    = -data->sin_phi * p.x + data->cos_phi * p.y;
            P.z                    = p.z;
            _P = RAYS_NORM( O, P );
            P.x = data->cos_phi * _P.x - data->sin_phi * _P.y;
            P.y = data->sin_phi * _P.x + data->cos_phi * _P.y;
            P.z = _P.z;
            return P;
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        rotacioQ,
        {
            primitives::bazo_ptr O = obj + data->o;
            matrix               Q;
            scalar               temp;
            
            Q.x.x = data->q.x * data->q.x;
            Q.y.y = data->q.y * data->q.y;
            Q.z.z = data->q.z * data->q.z;
            temp = Q.x.x + Q.y.y + Q.z.z;
            Q.x.x -= temp;
            Q.y.y -= temp;
            Q.z.z -= temp;
            
            Q.x.y = data->q.x * data->q.y;
            temp = data->q.z * data->q_w;
            Q.y.x = Q.x.y + temp;
            Q.x.y -= temp;
            
            Q.y.z = data->q.y * data->q.z;
            temp = data->q.x * data->q_w;
            Q.z.y = Q.y.z + temp;
            Q.y.z -= temp;
            
            Q.z.x = data->q.z * data->q.x;
            temp = data->q.y * data->q_w;
            Q.x.z = Q.z.x + temp;
            Q.z.x -= temp;
            
            point P = p;
            P.x += 2.f * ( Q.x.x * p.x + Q.x.y * p.y + Q.x.z * p.z );
            P.y += 2.f * ( Q.y.x * p.x + Q.y.y * p.y + Q.y.z * p.z );
            P.z += 2.f * ( Q.z.x * p.x + Q.z.y * p.y + Q.z.z * p.z );
            return RAYS_DIST( O, P );
        },
        {
            primitives::bazo_ptr O = obj + data->o;
            matrix               Q;
            scalar               temp;
            
            Q.x.x = data->q.x * data->q.x;
            Q.y.y = data->q.y * data->q.y;
            Q.z.z = data->q.z * data->q.z;
            temp = Q.x.x + Q.y.y + Q.z.z;
            Q.x.x -= temp;
            Q.y.y -= temp;
            Q.z.z -= temp;
            
            Q.x.y = data->q.x * data->q.y;
            temp = data->q.z * data->q_w;
            Q.y.x = Q.x.y + temp;
            Q.x.y -= temp;
            
            Q.y.z = data->q.y * data->q.z;
            temp = data->q.x * data->q_w;
            Q.z.y = Q.y.z + temp;
            Q.y.z -= temp;
            
            Q.z.x = data->q.z * data->q.x;
            temp = data->q.y * data->q_w;
            Q.x.z = Q.z.x + temp;
            Q.z.x -= temp;
            
            point P = p;
            P.x += 2.f * ( Q.x.x * p.x + Q.x.y * p.y + Q.x.z * p.z );
            P.y += 2.f * ( Q.y.x * p.x + Q.y.y * p.y + Q.y.z * p.z );
            P.z += 2.f * ( Q.z.x * p.x + Q.z.y * p.y + Q.z.z * p.z );
            point N = RAYS_NORM( O, P );
            P = N;
            P.x += 2.f * ( Q.x.x * N.x + Q.y.x * N.y + Q.z.x * N.z );
            P.y += 2.f * ( Q.x.y * N.x + Q.y.y * N.y + Q.z.y * N.z );
            P.z += 2.f * ( Q.x.z * N.x + Q.y.z * N.y + Q.z.z * N.z );
            return P;
        } );

CREATE_OBJECT_TYPE_DEFINITION(
        senfina_ripeto,
        {
            primitives::bazo_ptr o = obj + data->o;
            point                a = data->a;
            scalar               N = floorf( atomic::dot( a, p ) / atomic::dot( a, a ) + .5f );
            a.x = p.x - N * a.x;
            a.y = p.y - N * a.y;
            a.z = p.z - N * a.z;
            return RAYS_DIST( o, a );
        },
        {
            primitives::bazo_ptr o = obj + data->o;
            point                a = data->a;
            scalar               N = floorf( atomic::dot( a, p ) / atomic::dot( a, a ) + .5f );
            a.x                    = p.x - N * a.x;
            a.y                    = p.y - N * a.y;
            a.z                    = p.z - N * a.z;
            return RAYS_NORM( o, a );
        } );
    
};

// Kernels definitions

_KERNEL kernel_Process( const size_t *Width, const size_t *Height, const rays_info *Info_d, ray _PTR Rays,
                        primitives::bazo _PTR Primitives_d, const size_t *PrimitivesNum, hipSurfaceObject_t Image ) {
    coord
        x = CUDA_RAYS_COORD_nD( x, 2 ),
        y = CUDA_RAYS_COORD_nD( y, 2 ),
        id = RAYS_PRIMITIVES_PER_THREAD * ( threadIdx.y * RAYS_BLOCK_2D_x + threadIdx.x );

    // RAYS_BLOCK_2D_x * RAYS_BLOCK_2D_y * PRIMITIVES_PER_THREAD >= PrimitivesNum
    __shared__ primitives::bazo curr_ptr[ RAYS_BLOCK_2D_x * RAYS_BLOCK_2D_y * RAYS_PRIMITIVES_PER_THREAD ];
    if ( id < *PrimitivesNum ) {
        primitives::bazo_ptr self = curr_ptr + id;

#pragma unroll
        for ( size_t i = 0; i < RAYS_PRIMITIVES_PER_THREAD; ++i, ++self ) {
            *self = Primitives_d[ id + i ];
            //CREATE_OBJECT_TYPE_PROCESSING_LISTING_2( self );
        }
    }
    __syncthreads();

    if ( x < *Width && y < *Height ) {
        scalar curr_dist, ray_dist = 0;
        ray r = Rays[ y * *Width + x ];
        uchar4 PIXEL = { 0x00, 0x00, 0x00, 0xff };
        point curr_norm, light =  Info_d->LightSource; //point{ 1.f, 0.f, 0.f };

#pragma unroll
        for ( size_t I = 0; I < 500; ++I ) {
            curr_dist = RAYS_DIST( curr_ptr, r.p );

            if ( curr_dist < RAYS_MIN_DIST ) {
                if ( curr_dist < 0.f ) {
                    curr_norm.x = -r.d.x;
                    curr_norm.y = -r.d.y;
                    curr_norm.z = -r.d.z;
                } else {
                    curr_norm = RAYS_NORM( curr_ptr, r.p );
                }

                if ( atomic::dot( curr_norm, r.d ) < 0.f ) {
                    scalar R_1 = r_length_3( curr_norm.x, curr_norm.y, curr_norm.z ), N_L;
                    curr_norm = atomic::mul_point( curr_norm, R_1 );
                    N_L = atomic::dot( curr_norm, light );

                    float
                        SHADOW = 1.f,
                        OCCLUSION = 0.f,
                        SCA = 1.f;
                    point p = r.p;

                    ray_dist = RAYS_MIN_DIST;

//  #pragma unroll
//                      for ( size_t J = 0; J < 5; ++J ) {
//                          curr_dist = RAYS_DIST( curr_ptr, p );
//                          OCCLUSION += ( ray_dist - curr_dist ) * SCA;
//                          SCA *= .95;
//
//                          p.x += .04 * curr_dist * curr_norm.x;
//                          p.y += .04 * curr_dist * curr_norm.y;
//                          p.z += .04 * curr_dist * curr_norm.z;
//                          ray_dist += curr_dist;
//                      }
//                      OCCLUSION = ( 1.f - 1.5f * OCCLUSION );
//                      if ( OCCLUSION > 1.f )
//                          OCCLUSION = 1.f;
//                      if ( OCCLUSION < 0.f )
//                          OCCLUSION = 0.f;

#define DELTA       5
#define HARDNESS    128.f

                    p = r.p;
                    p.x += DELTA * RAYS_MIN_DIST * light.x;
                    p.y += DELTA * RAYS_MIN_DIST * light.y;
                    p.z += DELTA * RAYS_MIN_DIST * light.z;
                    ray_dist = DELTA * RAYS_MIN_DIST;

//#pragma unroll
                    for ( size_t J = 0; J < 200; ++J ) {
                        curr_dist = RAYS_DIST( curr_ptr, p );
                        
                        SHADOW = min( SHADOW, HARDNESS * curr_dist / ray_dist );
                        if ( SHADOW < 0.01f )
                            break;

                        p.x += curr_dist * light.x;
                        p.y += curr_dist * light.y;
                        p.z += curr_dist * light.z;
                        ray_dist += curr_dist;

                        // LIGHT
                        if ( ray_dist >= RAYS_MAX_DIST )
                            break;
                    }

                    float3 MATERIAL = { 1.f, 1.f, 1.f };
                    raw_byte LIGHT =
                        0xff * ( RAYS_MIN_LUM + ( RAYS_MAX_LUM - RAYS_MIN_LUM ) * ( N_L > 0.f ? N_L : 0.f ) * SHADOW );
                        //  0xff * ( RAYS_MIN_LUM + ( RAYS_MAX_LUM - RAYS_MIN_LUM ) * .5f * ( N_L + 1.f ) * SHADOW );
                    PIXEL = {
                         raw_byte( LIGHT * MATERIAL.x ),
                         raw_byte( LIGHT * MATERIAL.y ),
                         raw_byte( LIGHT * MATERIAL.z ),
                         0xff
                    };
                    break;
                }
            }

            r.p.x += curr_dist * r.d.x;
            r.p.y += curr_dist * r.d.y;
            r.p.z += curr_dist * r.d.z;
            ray_dist += curr_dist;

            if ( ray_dist >= RAYS_MAX_DIST ) {
                break;
            }
        }

        surf2Dwrite( PIXEL, Image, x * 4, y );
    }
}

_KERNEL kernel_SetPrimitives( primitives::bazo _PTR Primitives, const size_t *PrimitivesNum ) {
    coord x = CUDA_RAYS_COORD_nD( x, 1 );
    
    if ( x < *PrimitivesNum ) {
        primitives::bazo_ptr self = Primitives + x;
        CREATE_OBJECT_TYPE_PROCESSING_LISTING_2( self )
    }
}

_KERNEL kernel_SetRays( const size_t *Width, const size_t *Height, rays_info _PTR Info_d, ray _PTR Rays_d ) {
    coord
        x = CUDA_RAYS_COORD_nD( x, 2 ),
        y = CUDA_RAYS_COORD_nD( y, 2 );

    __shared__ rays_info Info[ 1 ];
    if ( threadIdx.x == 0 && threadIdx.y == 0 )
        Info[ 0 ] = *Info_d;
    __syncthreads();

    if ( x < *Width && y < *Height ) {
        scalar
            X = .5f * float( 2 * x - coord( *Width ) + 1 ),
            Y = .5f * float( 2 * y - coord( *Height ) + 1 ),
            Z = Info->Depth;

        point pos;
        pos.x = X * Info->StartWVec.x + Y * Info->StartHVec.x;
        pos.y = X * Info->StartWVec.y + Y * Info->StartHVec.y;
        pos.z = X * Info->StartWVec.z + Y * Info->StartHVec.z;

        point delta_pos;
        delta_pos.x = Z * Info->StartDir.x;
        delta_pos.y = Z * Info->StartDir.y;
        delta_pos.z = Z * Info->StartDir.z;

        scalar R_1 = rnorm3df( pos.x + delta_pos.x, pos.y + delta_pos.y, pos.z + delta_pos.z );

        ray *self = Rays_d + y * *Width + x;
        self->d = atomic::mul_point( atomic::add_point( pos, delta_pos ), R_1 );
        self->p = atomic::add_point( pos, Info->StartPos );
    }
}

#include "../Include/cuda_rays.cuh"

#define _CUDA(__ERROR__)    {_last_cuda_error = hipError_t(__ERROR__); CUDA_CHECK(_last_cuda_error);}
#define _NVRTC(__ERROR__)   {_last_nvrtc_error = __ERROR__; }
#define _RETURN             return int(_last_cuda_error);

#define grid_1d(X)          (dim3( ( (X) - 1 ) / RAYS_BLOCK_1D_x + 1 ))
#define grid_2d(X,Y)        (dim3( ( (X) - 1 ) / RAYS_BLOCK_2D_x + 1, ( (Y) - 1 ) / RAYS_BLOCK_2D_y + 1 ))

#define block_1d            (dim3((RAYS_BLOCK_1D_x)))
#define block_2d            (dim3((RAYS_BLOCK_2D_x),(RAYS_BLOCK_2D_y)))

#define _KERNEL(__FUNC__)    _CUDA(hipModuleLaunchKernel((__FUNC__),grid.x,grid.y,grid.z,block.x,block.y,block.z,0,_default_stream,args,nullptr))

namespace cuda {

int raymarching::Init( rays_Init_args ) {
    _CUDA( hipInit( 0 ) )
    
    //  Only one device
    _CUDA( hipDeviceGet( &_device, 0 ) )
    int cc[2];
    _CUDA( hipDeviceComputeCapability( cc, cc + 1, _device ) )
    _cc_div_10 = 10 * cc[0] + cc[1];
    _CUDA( hipDeviceGetName( _device_name, 128, _device ) )
    
    _CUDA( hipCtxCreate( &_context, 0, _device ) )
    
    size_t          _cubin_len;
    void*           _cubin_src;
    
    {
        // Create build environment
        std::string         _cuda_source =
                IO::read_source( std::string(__PROJ_DIR__) + "Source/cuda_kernels.cu" );
        hiprtcProgram        _kernel;
        _NVRTC( hiprtcCreateProgram(
                &_kernel,
                _cuda_source.c_str(),
                "cudaRayMarching",
                0, nullptr, nullptr ) )
        
        _NVRTC( hiprtcAddNameExpression( _kernel, "kernel_Process" ) )
        _NVRTC( hiprtcAddNameExpression( _kernel, "kernel_SetPrimitives" ) )
        _NVRTC( hiprtcAddNameExpression( _kernel, "kernel_SetRays" ) )
        
        const std::string _arch_flag = "-arch=compute_" + std::to_string(_cc_div_10);
        const char*     _options[] = {
                _arch_flag.c_str(),
                "-use_fast_math",
                "-dc",
                "-std=c++17",
                "-builtin-initializer-list=true",
                "-I./Include",
        };
        _NVRTC( hiprtcCompileProgram( _kernel, 6, _options ) )
        
        size_t          _nvrtc_log_len;
        _NVRTC( hiprtcGetProgramLogSize( _kernel, &_nvrtc_log_len ) )
        char*           _nvrtc_log_src = new char [ _nvrtc_log_len ];
        _NVRTC( hiprtcGetProgramLog( _kernel, _nvrtc_log_src ))
        
        size_t          _ptx_len;
        _NVRTC( hiprtcGetCodeSize( _kernel, &_ptx_len ) )
        char*           _ptx_src = new char [ _ptx_len ];
        _NVRTC( hiprtcGetCode( _kernel, _ptx_src ) )
        
        size_t          _jit_info_log_len = 1 << 14,
                        _jit_err_log_len = 1 << 14;
        char            *_jit_info_log_src = new char [ _jit_info_log_len ],
                        *_jit_err_log_src = new char [ _jit_err_log_len ];
        
        size_t          _jit_options_count = 5;
        hipJitOption    _jit_options_list[] = {
                hipJitOptionTarget,
                hipJitOptionInfoLogBufferSizeBytes,
                hipJitOptionErrorLogBufferSizeBytes,
                hipJitOptionInfoLogBuffer,
                hipJitOptionErrorLogBuffer,
        };
        void*           _jit_options_value[] = {
                (void*) _cc_div_10,
                (void*) _jit_info_log_len,
                (void*) _jit_err_log_len,
                (void*) _jit_info_log_src,
                (void*) _jit_err_log_src,
        };
        
        _CUDA( hiprtcLinkCreate(
                _jit_options_count, _jit_options_list, _jit_options_value,
                &_link_state ) )
        _CUDA( hiprtcLinkAddFile(
                _link_state, hipJitInputLibrary,
                CUDA_LIB_PATH_WIN(cudadevrt),
                0, nullptr, nullptr ) )
        _CUDA( hiprtcLinkAddData(
                _link_state, hipJitInputPtx,
                _ptx_src, _ptx_len, "cuda_kernel.ptx",
                0, nullptr, nullptr ) )
        _CUDA( hiprtcLinkComplete( _link_state, &_cubin_src, &_cubin_len ) )
        
        // Cleaning build environment
        _NVRTC( hiprtcDestroyProgram( &_kernel ) )
        delete[]        _ptx_src;
        delete[]        _nvrtc_log_src;
        delete[]        _jit_info_log_src;
        delete[]        _jit_err_log_src;
        
        auto *_file_name = new std::string(__PROJ_DIR__);
        *_file_name += "cuda_kernel_" + std::to_string(_cc_div_10) + ".cubin";
        IO::write_binary_nowait( _file_name, _cubin_src, _cubin_len );
    }
    
    _CUDA( hipModuleLoadData( &_module, _cubin_src ) )
    
    //_CUDA( hipModuleLoad( &_module, _PATH ) )
    _CUDA( hipModuleGetFunction( &_process,          _module, "kernel_Process" ) )
    _CUDA( hipModuleGetFunction( &_set_primitives,   _module, "kernel_SetPrimitives" ) )
    _CUDA( hipModuleGetFunction( &_set_rays,         _module, "kernel_SetRays" ) )
    
    for ( size_t n = 0; n < CUDA_RAYS_STREAM_NUM; ++n ) {
        _CUDA( hipStreamCreateWithFlags( _stream + n, 0 ) )
    }
    _default_stream = _stream[ CUDA_RAYS_DEFAULT_STREAM ];
    
    for ( size_t n = 0; n < CUDA_RAYS_EVENT_NUM; ++n ) {
        _CUDA( hipEventCreateWithFlags( _event + n, 0 ) )
    }
    
    Width = width;
    Height = height;
    
    _CUDA( hipMalloc( &_rays, Width * Height * sizeof( ray ) ) )
    _CUDA( hipMalloc( &_info, sizeof( rays_info ) ) )
    _prim = 0;
    
    _CUDA( hipMalloc( &_width, sizeof( size_t ) ) )
    _CUDA( hipMalloc( &_height, sizeof( size_t ) ) )
    _CUDA( hipMalloc( &_prim_num, sizeof( size_t ) ) )
    
    _resource = nullptr;
    
    _resource_desc.flags = 0;
    _resource_desc.resType = HIP_RESOURCE_TYPE_ARRAY;
    
    _RETURN
}

int raymarching::Process( rays_Process_args ) {
    _CUDA( hipEventRecord( _event[0], _default_stream ) )
    
    static dim3 grid = grid_2d( Width, Height ), block = block_2d;
    static void* args[] = { &_width, &_height, &_info, &_rays, &_prim, &_prim_num, &_surface };
    _KERNEL( _process )
    
    //  kernel::Process <<< grid_2d( Width, Height ), block_2d, 0, _default_stream >>>
    //      ( Width, Height, _INFO, _RAYS, _PRIM, PrimitivesNum, _surface );
    
    _CUDA( hipEventRecord( _event[ 1 ], _default_stream ) )
    _CUDA( hipStreamSynchronize( _default_stream ) )
    _CUDA( hipEventElapsedTime( &_last_process_time, _event[ 0 ], _event[ 1 ] ) )
    
    return uint32_t( _last_process_time );
}

int raymarching::Quit( rays_Quit_args ) {
    if (_prim) _CUDA( hipFree( _prim ) )
    if (_rays) _CUDA( hipFree( _rays ) )
    if (_info) _CUDA( hipFree( _info ) )
    
    _CUDA( hipModuleUnload( _module ) )
    _CUDA( hipCtxDestroy( _context ) )
    
    _RETURN
}


int raymarching::SetInfo( rays_SetInfo_args ) {
    _CUDA( hipMemcpyHtoDAsync( _info, &info, sizeof( rays_info ), _default_stream ) )
    _CUDA( hipMemcpyHtoDAsync( _width, &Width, sizeof( size_t ), _default_stream ) )
    _CUDA( hipMemcpyHtoDAsync( _height, &Height, sizeof( size_t ), _default_stream ) )
    _CUDA( SetRays() )
    _CUDA( hipStreamSynchronize( _default_stream ) )

    
    _RETURN
}

int raymarching::SetTexture( rays_SetTexture_args ) {
    if ( _resource ) {
        _CUDA( UnsetTexture() )
    }
    
    _CUDA( hipGraphicsGLRegisterImage( &_resource, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore ) )
    _CUDA( hipGraphicsMapResources( 1, &_resource, _default_stream ) )
    
    _CUDA( hipGraphicsSubResourceGetMappedArray( &_resource_desc.res.array.hArray, _resource, 0, 0 ) )
    
    _CUDA( cuSurfObjectCreate( &_surface, &_resource_desc ) )
    
    _RETURN
}

int raymarching::UnsetTexture( rays_UnsetInfo_args ) {
    _CUDA( cuSurfObjectDestroy( _surface ) )
    _CUDA( hipGraphicsUnmapResources( 1, &_resource, _default_stream ) )
    _CUDA( hipGraphicsUnregisterResource( _resource ) )
    
    _resource = nullptr;
    
    _RETURN
}

int raymarching::SetPrimitives( rays_SetPrimitives_args ) {
    PrimitivesNum = Primitives_h.size();
    
    if ( _prim ) {
        _CUDA( hipFree( _prim ) )
    }
    
    _CUDA( hipMalloc( &_prim, PrimitivesNum * sizeof( primitives::bazo )  ) )
    _CUDA( hipMemcpyHtoDAsync( _prim, Primitives_h.data(), PrimitivesNum * sizeof( primitives::bazo ), _default_stream ) )
    _CUDA( hipMemcpyHtoDAsync( _prim_num, &PrimitivesNum, sizeof( size_t ), _default_stream ) )
    
    static dim3 grid = grid_1d( PrimitivesNum ), block = block_1d;
    static void* args[] = { &_prim, &_prim_num };
    _KERNEL( _set_primitives )
    
    //  kernel::SetPrimitives <<< grid_1d( PrimitivesNum ), block_1d, 0, _default_stream >>>
    //      ( _PRIM, PrimitivesNum );
    
    _CUDA( hipStreamSynchronize( _default_stream ) )
    
    _RETURN
}

int raymarching::SetRays( rays_SetRays_args ) {
    static dim3 grid = grid_2d( Width, Height ), block = block_2d;
    static void* args[] = { &_width, &_height, &_info, &_rays };
    _KERNEL( _set_rays )
    
    //  kernel::SetRays <<< grid_2d( Width, Height ), block_2d, 0, _default_stream >>>
    //      ( Width, Height, _INFO, _RAYS );
    
    _RETURN
}

};
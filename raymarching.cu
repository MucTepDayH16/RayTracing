#include "hip/hip_runtime.h"
#include "rays.h"
#include <cuda_device_runtime_api.h>

namespace primitives {

__device__ __forceinline__ point Point( scalar x, scalar y, scalar z ) {
    return point{ x, y, z };
}

__device__ __forceinline__ scalar mix( scalar a, scalar b, scalar x ) {
    return b + ( a - b ) * x;
}

__device__ __forceinline__ scalar dot( point a, point b ) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

// TYPE_LIST
CREATE_OBJECT_TYPE_DEFINITION(
    portanta_sfero,
    {
        point P;
        P.x = p.x - data->t.x;
        P.y = p.y - data->t.y;
        P.z = p.z - data->t.z;
        scalar d = length_3( P.x, P.y, P.z ) - data->r;
        bazo_ptr o = obj + data->o;
        if ( d <= RAYS_MIN_DIST )   return RAYS_DIST( o, P );
        else                        return d;
    },
    {
        point P;
        P.x = p.x - data->t.x;
        P.y = p.y - data->t.y;
        P.z = p.z - data->t.z;
        bazo_ptr o = obj + data->o;
        return RAYS_NORM( o, P );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    sfero,
    {
        return length_3( p.x, p.y, p.z ) - data->r;
    },
    {
        return p;
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    kubo,
    {
        point q;
        q.x = fabsf( p.x ) - data->b.x;
        q.y = fabsf( p.y ) - data->b.y;
        q.z = fabsf( p.z ) - data->b.z;
        if ( q.x < 0.f && q.y < 0.f && q.z < 0.f )
            return max( q.x, max( q.y, q.z ) );
        else
            return length_3( max( q.x, 0.f ), max( q.y, 0.f ), max( q.z, 0.f ) );
    },
    {
        point q;
        q.x = fabsf( p.x ) - data->b.x;
        q.y = fabsf( p.y ) - data->b.y;
        q.z = fabsf( p.z ) - data->b.z;
        if ( q.x < 0.f && q.y < 0.f && q.z < 0.f )
            return q.x > q.z ? ( q.x > q.y ? Point( p.x > 0.f ? 1.f : -1.f, 0.f, 0.f ) : Point( 0.f, p.y > 0.f ? 1.f : -1.f, 0.f ) ) : ( q.y > q.z ? Point( 0.f, p.y > 0.f ? 1.f : -1.f, 0.f ) : Point( 0.f, 0.f, p.z > 0.f ? 1.f : -1.f ) );
        else
            return q.x > q.z ? ( q.x > q.y ? Point( p.x > 0.f ? 1.f : -1.f, 0.f, 0.f ) : Point( 0.f, p.y > 0.f ? 1.f : -1.f, 0.f ) ) : ( q.y > q.z ? Point( 0.f, p.y > 0.f ? 1.f : -1.f, 0.f ) : Point( 0.f, 0.f, p.z > 0.f ? 1.f : -1.f ) );
        //Point( q.x > 0.f ? ( p.x >= data->c.x ? 1.f : -1.f ) : 0.f, q.y > 0.f ? ( p.y >= data->c.y ? 1.f : -1.f ) : 0.f, q.z > 0.f ? ( p.z >= data->c.z ? 1.f : -1.f ) : 0.f );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    cilindro,
    {
        scalar r = length_2( p.x, p.y );
        float2 q;
        q.x = r - data->r;
        q.y = fabsf( p.z ) - data->h;
        if ( q.x < 0.f && q.y < 0.f )
            return q.x > q.y ? q.x : q.y;
        else
            return length_2( max( q.x, 0.f ), max( q.y, 0.f ) );
    },
    {
        scalar r = length_2( p.x, p.y );
        float2 q;
        q.x = r - data->r;
        q.y = fabsf( p.z ) - data->h;
        if ( q.x < 0.f && q.y < 0.f )
            return q.x > q.y ? Point( p.x, p.y, 0.f ) : Point( 0.f, 0.f, p.z > 0.f ? 1.f : -1.f );
        else
            return q.x > q.y ? Point( p.x, p.y, 0.f ) : Point( 0.f, 0.f, p.z > 0.f ? 1.f : -1.f );

    } );


CREATE_OBJECT_TYPE_DEFINITION(
    kunigajo_2,
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );
        return min( d0, d1 );
    },
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );

        if ( d0 < d1 )  return RAYS_NORM( o0, p );
        else            return RAYS_NORM( o1, p );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    kunigajo_3,
    {
        bazo_ptr o = obj + data->o[ 0 ];
        scalar d = RAYS_DIST( o, p );

        o = obj + data->o[ 1 ];
        d = min( d, RAYS_DIST( o, p ) );

        o = obj + data->o[ 2 ];
        return min( d, RAYS_DIST( o, p ) );
    },
    {
        counter i_min = 0;
        bazo_ptr o = obj + data->o[ 0 ];
        scalar d; scalar d_min = RAYS_DIST( o, p );

        o = obj + data->o[ 1 ];
        d = RAYS_DIST( o, p );
        if ( d_min > d ) { d_min = d; i_min = 1; }

        o = obj + data->o[ 2 ];
        d = RAYS_DIST( o, p );
        if ( d_min > d ) { i_min = 2; }

        o = obj + data->o[ i_min ];
        return RAYS_NORM( o, p );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    kunigajo_4,
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );
        return min( d0, d1 );
    },
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );

        if ( d0 < d1 )  return RAYS_NORM( o0, p );
        else            return RAYS_NORM( o1, p );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    komunajo_2,
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );
        return max( d0, d1 );
    },
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );

        if ( d0 > d1 )  return RAYS_NORM( o0, p );
        else            return RAYS_NORM( o1, p );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    komunajo_3,
    {
        bazo_ptr o = obj + data->o[ 0 ];
        scalar d = RAYS_DIST( o, p );

        o = obj + data->o[ 1 ];
        d = max( d, RAYS_DIST( o, p ) );

        o = obj + data->o[ 2 ];
        return max( d, RAYS_DIST( o, p ) );
    },
    {
        counter i_max = 0;
        bazo_ptr o = obj + data->o[ 0 ];
        scalar d; scalar d_max = RAYS_DIST( o, p );

        o = obj + data->o[ 1 ];
        d = RAYS_DIST( o, p );
        if ( d_max < d ) { d_max = d; i_max = 1; }

        o = obj + data->o[ 2 ];
        d = RAYS_DIST( o, p );
        if ( d_max < d ) { i_max = 2; }

        o = obj + data->o[ i_max ];
        return RAYS_NORM( o, p );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    komunajo_4,
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );
        return max( d0, d1 );
    },
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );

        if ( d0 > d1 )  return RAYS_NORM( o0, p );
        else            return RAYS_NORM( o1, p );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    komplemento,
    {
        bazo_ptr O = obj + data->o;
        scalar D = RAYS_DIST( O, p );
        return -D;
    },
    {
        bazo_ptr O = obj + data->o;
        point N = RAYS_NORM( O, p );
        return Point( -N.x, -N.y, -N.z );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    glata_kunigajo_2,
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );
        scalar h = ( 1.f - ( d0 - d1 ) / data->k ) * .5f;
        if ( h > 1.f )  return d0;
        if ( h < 0.f )  return d1;
        return mix( d0, d1, h ) - data->k * h * ( 1.f - h );
    },
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );
        scalar h = ( 1.f - ( d0 - d1 ) / data->k ) * .5f;
        if ( h > 1.f )  return RAYS_NORM( o0, p );
        if ( h < 0.f )  return RAYS_NORM( o1, p );
        point n0 = RAYS_NORM( o0, p ); point n1 = RAYS_NORM( o1, p );
        d0 = r_length_3( n0.x, n0.y, n0.z );
        d1 = r_length_3( n1.x, n1.y, n1.z );
        return Point( mix( d0 * n0.x, d1 * n1.x, h ), mix( d0 * n0.y, d1 * n1.y, h ), mix( d0 * n0.z, d1 * n1.z, h ) );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    glata_komunajo_2,
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );
        scalar h = ( 1.f + ( d0 - d1 ) / data->k ) * .5f;
        if ( h > 1.f )  return d0;
        if ( h < 0.f )  return d1;
        return mix( d0, d1, h ) + data->k * h * ( 1.f - h );
    },
    {
        bazo_ptr o0 = obj + data->o[ 0 ]; bazo_ptr o1 = obj + data->o[ 1 ];
        scalar d0 = RAYS_DIST( o0, p ); scalar d1 = RAYS_DIST( o1, p );
        scalar h = ( 1.f + ( d0 - d1 ) / data->k ) * .5f;
        if ( h > 1.f )  return RAYS_NORM( o0, p );
        if ( h < 0.f )  return RAYS_NORM( o1, p );
        point n0 = RAYS_NORM( o0, p ); point n1 = RAYS_NORM( o1, p );
        d0 = r_length_3( n0.x, n0.y, n0.z );
        d1 = r_length_3( n1.x, n1.y, n1.z );
        return Point( mix( d0 * n0.x, d1 * n1.x, h ), mix( d0 * n0.y, d1 * n1.y, h ), mix( d0 * n0.z, d1 * n1.z, h ) );
    } );


CREATE_OBJECT_TYPE_DEFINITION(
    movo,
    {
        bazo_ptr O = obj + data->o;
        point P;
        P.x = p.x - data->t.x;
        P.y = p.y - data->t.y;
        P.z = p.z - data->t.z;
        return RAYS_DIST( O, P );
    },
    {
        bazo_ptr O = obj + data->o;
        point P;
        P.x = p.x - data->t.x;
        P.y = p.y - data->t.y;
        P.z = p.z - data->t.z;
        return RAYS_NORM( O, P );
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    rotacioX,
    {
        bazo_ptr O = obj + data->o;
        point P;
        P.y = data->cos_phi * p.y + data->sin_phi * p.z;
        P.z = -data->sin_phi * p.y + data->cos_phi * p.z;
        P.x = p.x;
        return RAYS_DIST( O, P );
    },
    {
        bazo_ptr O = obj + data->o;
        point P; point _P;
        P.y = data->cos_phi * p.y + data->sin_phi * p.z;
        P.z = -data->sin_phi * p.y + data->cos_phi * p.z;
        P.x = p.x;
        _P = RAYS_NORM( O, P );
        P.y = data->cos_phi * _P.y - data->sin_phi * _P.z;
        P.z = data->sin_phi * _P.y + data->cos_phi * _P.z;
        P.x = _P.x;
        return P;
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    rotacioY,
    {
        bazo_ptr O = obj + data->o;
        point P;
        P.z = data->cos_phi * p.z + data->sin_phi * p.x;
        P.x = -data->sin_phi * p.z + data->cos_phi * p.x;
        P.y = p.y;
        return RAYS_DIST( O, P );
    },
    {
        bazo_ptr O = obj + data->o;
        point P; point _P;
        P.z = data->cos_phi * p.z + data->sin_phi * p.x;
        P.x = -data->sin_phi * p.z + data->cos_phi * p.x;
        P.y = p.y;
        _P = RAYS_NORM( O, P );
        P.z = data->cos_phi * _P.z - data->sin_phi * _P.x;
        P.x = data->sin_phi * _P.z + data->cos_phi * _P.x;
        P.y = _P.y;
        return P;
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    rotacioZ,
    {
        bazo_ptr O = obj + data->o;
        point P;
        P.x = data->cos_phi * p.x + data->sin_phi * p.y;
        P.y = -data->sin_phi * p.x + data->cos_phi * p.y;
        P.z = p.z;
        return RAYS_DIST( O, P );
    },
    {
        bazo_ptr O = obj + data->o;
        point P; point _P;
        P.x = data->cos_phi * p.x + data->sin_phi * p.y;
        P.y = -data->sin_phi * p.x + data->cos_phi * p.y;
        P.z = p.z;
        _P = RAYS_NORM( O, P );
        P.x = data->cos_phi * _P.x - data->sin_phi * _P.y;
        P.y = data->sin_phi * _P.x + data->cos_phi * _P.y;
        P.z = _P.z;
        return P;
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    rotacioQ,
    {
        bazo_ptr O = obj + data->o;
        matrix Q; scalar temp;

        Q.x.x = data->q.x * data->q.x;
        Q.y.y = data->q.y * data->q.y;
        Q.z.z = data->q.z * data->q.z;
        temp = Q.x.x + Q.y.y + Q.z.z;
        Q.x.x -= temp;
        Q.y.y -= temp;
        Q.z.z -= temp;

        Q.x.y = data->q.x * data->q.y;
        temp = data->q.z * data->q_w;
        Q.y.x = Q.x.y + temp;
        Q.x.y -= temp;

        Q.y.z = data->q.y * data->q.z;
        temp = data->q.x * data->q_w;
        Q.z.y = Q.y.z + temp;
        Q.y.z -= temp;

        Q.z.x = data->q.z * data->q.x;
        temp = data->q.y * data->q_w;
        Q.x.z = Q.z.x + temp;
        Q.z.x -= temp;

        point P = p;
        P.x += 2.f * ( Q.x.x * p.x + Q.x.y * p.y + Q.x.z * p.z );
        P.y += 2.f * ( Q.y.x * p.x + Q.y.y * p.y + Q.y.z * p.z );
        P.z += 2.f * ( Q.z.x * p.x + Q.z.y * p.y + Q.z.z * p.z );
        return RAYS_DIST( O, P );
    },
    {
        bazo_ptr O = obj + data->o;
        matrix Q; scalar temp;

        Q.x.x = data->q.x * data->q.x;
        Q.y.y = data->q.y * data->q.y;
        Q.z.z = data->q.z * data->q.z;
        temp = Q.x.x + Q.y.y + Q.z.z;
        Q.x.x -= temp;
        Q.y.y -= temp;
        Q.z.z -= temp;

        Q.x.y = data->q.x * data->q.y;
        temp = data->q.z * data->q_w;
        Q.y.x = Q.x.y + temp;
        Q.x.y -= temp;

        Q.y.z = data->q.y * data->q.z;
        temp = data->q.x * data->q_w;
        Q.z.y = Q.y.z + temp;
        Q.y.z -= temp;

        Q.z.x = data->q.z * data->q.x;
        temp = data->q.y * data->q_w;
        Q.x.z = Q.z.x + temp;
        Q.z.x -= temp;

        point P = p;
        P.x += 2.f * ( Q.x.x * p.x + Q.x.y * p.y + Q.x.z * p.z );
        P.y += 2.f * ( Q.y.x * p.x + Q.y.y * p.y + Q.y.z * p.z );
        P.z += 2.f * ( Q.z.x * p.x + Q.z.y * p.y + Q.z.z * p.z );
        point N = RAYS_NORM( O, P );
        P = N;
        P.x += 2.f * ( Q.x.x * N.x + Q.y.x * N.y + Q.z.x * N.z );
        P.y += 2.f * ( Q.x.y * N.x + Q.y.y * N.y + Q.z.y * N.z );
        P.z += 2.f * ( Q.x.z * N.x + Q.y.z * N.y + Q.z.z * N.z );
        return P;
    } );
CREATE_OBJECT_TYPE_DEFINITION(
    senfina_ripeto,
    {
        bazo_ptr o = obj + data->o;
        point a = data->a; scalar N = floorf( dot( a, p ) / dot( a, a ) + .5f );
        a.x = p.x - N * a.x;
        a.y = p.y - N * a.y;
        a.z = p.z - N * a.z;
        return RAYS_DIST( o, a );
    },
    {
        bazo_ptr o = obj + data->o;
        point a = data->a; scalar N = floorf( dot( a, p ) / dot( a, a ) + .5f );
        a.x = p.x - N * a.x;
        a.y = p.y - N * a.y;
        a.z = p.z - N * a.z;
        return RAYS_NORM( o, a );
    } );
};

namespace raymarching {

static size_t Width, Height;
static hipSurfaceObject_t Surface_d;

static point *LightSource_d;
static primitives::bazo *Primitives_d;
static size_t PrimitivesNum;
static ray *Rays_d;
static start_init_rays_info *Info_d;


static dim3 block_1d( RAYS_BLOCK_1D_x );
static dim3 block_2d( RAYS_BLOCK_2D_x, RAYS_BLOCK_2D_y );
static dim3 block_3d( RAYS_BLOCK_3D_x, RAYS_BLOCK_3D_y, RAYS_BLOCK_3D_z );

static dim3 grid( size_t X ) {
    return dim3( ( X - 1 ) / RAYS_BLOCK_1D_x + 1 );
}

static dim3 grid( size_t X, size_t Y ) {
    return dim3( ( X - 1 ) / RAYS_BLOCK_2D_x + 1, ( Y - 1 ) / RAYS_BLOCK_2D_y + 1 );
}

static dim3 grid( size_t X, size_t Y, size_t Z ) {
    return dim3( ( X - 1 ) / RAYS_BLOCK_3D_x + 1, ( Y - 1 ) / RAYS_BLOCK_3D_y + 1, ( Z - 1 ) / RAYS_BLOCK_3D_z + 1 );
}

static __device__ __inline__ point mul_point( const point &p, const scalar &s ) {
    return point{ s * p.x, s * p.y, s * p.z };
}

static __device__ __inline__ point add_point( const point &p1, const point &p2 ) {
    return point{ p1.x + p2.x, p1.y + p2.y, p1.z + p2.z };
}

static __device__ __inline__ scalar dot( const point &p1, const point &p2 ) {
    return p1.x * p2.x + p1.y * p2.y + p1.z * p2.z;
}

int Init( size_t width, size_t height, const hipSurfaceObject_t &surface ) {
    Width = width;
    Height = height;
    Surface_d = surface;

    CUDA_ERROR( hipMalloc( &LightSource_d, sizeof point ) );
    CUDA_ERROR( hipMalloc( &Rays_d, Width * Height * sizeof ray ) );
    CUDA_ERROR( hipMalloc( &Info_d, sizeof start_init_rays_info ) );
    return 1;
}

static __global__ void kernelInitPrimitives( primitives::bazo KERNEL_PTR Primitives, size_t PrimitivesNum ) {
    size_t x = RAYS_COORD_nD( x, 1 );

    if ( x < PrimitivesNum ) {
        primitives::bazo_ptr self = Primitives + x;
        CREATE_OBJECT_TYPE_PROCESSING_LISTING_2( self );
    }
}

int InitPrimitives( std::list< primitives::bazo_ptr > &Primitives, hipStream_t stream ) {
    PrimitivesNum = Primitives.size();

    if ( Primitives_d ) CUDA_ERROR( hipFree( Primitives_d ) );
    CUDA_ERROR( hipMalloc( &Primitives_d, PrimitivesNum * sizeof primitives::bazo ) );

    size_t i = 0;
    for ( primitives::bazo_ptr ptr : Primitives ) {
        CUDA_ERROR( hipMemcpyAsync( Primitives_d + i, ptr, sizeof primitives::bazo, hipMemcpyHostToDevice, stream ) );
        ++i;
    }

    kernelInitPrimitives <<< grid( PrimitivesNum ), block_1d, 0, stream >>> ( Primitives_d, PrimitivesNum );

    return 1;
}

static __global__ void kernelLoad( start_init_rays_info KERNEL_PTR Info_d, ray KERNEL_PTR Rays ) {
    int64_t
        x = RAYS_COORD_nD( x, 2 ),
        y = RAYS_COORD_nD( y, 2 );

    __shared__ start_init_rays_info Info[ 1 ];
    if ( threadIdx.x == 0 && threadIdx.y == 0 )
        Info[ 0 ] = *Info_d;
    __syncthreads();

    if ( x < Info->Width && y < Info->Height ) {
        scalar
            X = .5f * ( 2 * x - int64_t( Info->Width ) + 1 ),
            Y = .5f * ( 2 * y - int64_t( Info->Height ) + 1 ),
            Z = Info->Depth;

        point pos;
        pos.x = X * Info->StartWVec.x + Y * Info->StartHVec.x;
        pos.y = X * Info->StartWVec.y + Y * Info->StartHVec.y;
        pos.z = X * Info->StartWVec.z + Y * Info->StartHVec.z;

        point delta_pos;
        delta_pos.x = Z * Info->StartDir.x;
        delta_pos.y = Z * Info->StartDir.y;
        delta_pos.z = Z * Info->StartDir.z;

        scalar R_1 = rnorm3df( pos.x + delta_pos.x, pos.y + delta_pos.y, pos.z + delta_pos.z );

        ray *self = Rays + y * Info->Width + x;
        self->d =
            //Info->StartDir;
            mul_point( add_point( pos, delta_pos ), R_1 );
        self->p = add_point( pos, Info->StartPos );
    }
}

int Load( point &LightSource, start_init_rays_info &Info, hipStream_t stream ) {
    CUDA_ERROR( hipMemcpyAsync( LightSource_d, &LightSource, sizeof point, hipMemcpyHostToDevice, stream ) );

    CUDA_ERROR( hipMemcpyAsync( Info_d, &Info, sizeof start_init_rays_info, hipMemcpyHostToDevice, stream ) );

    kernelLoad <<< grid( Width, Height ), block_2d, 0, stream >>> ( Info_d, Rays_d );
    CUDA_ERROR( hipStreamSynchronize( stream ) );

    return 1;
}

#define PRIMITIVES_PER_THREAD 2

static __global__ void kernelImageProcessing( hipSurfaceObject_t image, size_t width, size_t height, size_t time, ray KERNEL_PTR Rays, point KERNEL_PTR LightSource, primitives::bazo KERNEL_PTR Primitives, size_t PrimitivesNum ) {
    size_t  x = RAYS_COORD_nD( x, 2 ),
            y = RAYS_COORD_nD( y, 2 ),
            id = PRIMITIVES_PER_THREAD * ( threadIdx.y * RAYS_BLOCK_2D_x + threadIdx.x );

    // RAYS_BLOCK_2D_x * RAYS_BLOCK_2D_y * PRIMITIVES_PER_THREAD >= PrimitivesNum
    __shared__ primitives::bazo curr_ptr[ RAYS_BLOCK_2D_x * RAYS_BLOCK_2D_y * PRIMITIVES_PER_THREAD ];
    if ( id < PrimitivesNum ) {
        primitives::bazo_ptr self = curr_ptr + id;

#pragma unroll
        for ( uint16_t i = 0; i < PRIMITIVES_PER_THREAD; ++i, ++self ) {
            *self = Primitives[ id + i ];
            //CREATE_OBJECT_TYPE_PROCESSING_LISTING_2( self );
        }
    }
    __syncthreads();

    if ( x < width && y < height ) {
        scalar curr_dist, ray_dist = 0;
        ray r = Rays[ y * width + x ];

        for ( size_t I = 0; true; ++I ) {
            curr_dist = RAYS_DIST( curr_ptr, r.p );

            r.p.x += curr_dist * r.d.x;
            r.p.y += curr_dist * r.d.y;
            r.p.z += curr_dist * r.d.z;

            if ( curr_dist < RAYS_MIN_DIST ) {
                point curr_norm, light = *LightSource;
                if ( curr_dist < 0.f ) {
                    curr_norm.x = -r.d.x;
                    curr_norm.y = -r.d.y;
                    curr_norm.z = -r.d.z;
                } else {
                    curr_norm = RAYS_NORM( curr_ptr, r.p );
                }

                if ( dot( curr_norm, r.d ) < 0.f ) {
                    scalar R_1 = r_length_3( curr_norm.x, curr_norm.y, curr_norm.z );

                    uint8_t LIGHT = 0xff * ( RAYS_MIN_LUM + .5f * ( RAYS_MAX_LUM - RAYS_MIN_LUM ) * ( 1.f + R_1 * dot( curr_norm, light ) ) );
                    uchar4 PIXEL = { LIGHT, LIGHT, LIGHT, 0xff };
                    surf2Dwrite( PIXEL, image, x * 4, y );
                    break;
                }
            }

            ray_dist += curr_dist;

            if ( ray_dist > RAYS_MAX_DIST || I >= RAYS_MAX_COUNTER ) {
                uchar3 COLOR = uchar3{ 0x00, 0x00, 0x00 };
                uchar4 PIXEL = RGB_PIXEL( COLOR );
                surf2Dwrite( PIXEL, image, x * 4, y );
                break;
            }
        }
    }
}

bool ImageProcessing( size_t time, hipStream_t stream ) {
    kernelImageProcessing <<< grid( Width, Height ), block_2d, 0, stream >>>
        ( Surface_d, Width, Height, time, Rays_d, LightSource_d, Primitives_d, PrimitivesNum );
    CUDA_ERROR( hipStreamSynchronize( stream ) );
    return true;
}

bool Quit() {
    hipFree( Primitives_d );
    hipFree( LightSource_d );
    hipFree( Info_d );
    hipFree( Rays_d );
    return true;
}

};